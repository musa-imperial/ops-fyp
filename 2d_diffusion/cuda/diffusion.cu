
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define IDX(I, J) ((J)*Nx+ (I))

hipError_t cudaCheck(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

// CUDA diffusion kernel
__global__ void interior_kernel(double *Anew, double *A, int Npts, int Nx, double hnudt) 
{
    //Get our global thread ID 1D block and grid
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    //at id we are at (i, j)
    //id+1 -> (i+1, j)
    //id+blockIdx

    // Add for n number of elements
    if (id < Npts) 
    {
        if (id%Nx!=0 && (id+1)%Nx!=0 && id<Npts-Nx && id>Nx-1)
        Anew[id] = A[id]+hnudt*(A[id+1]+A[id-1]+A[id+Nx]+A[id-Nx]-4*A[id]);
    }
}

__global__ void copy(double *Anew, double *A, int Npts) {

    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < Npts) {
        A[id] = Anew[id];
    }
}

int main()
{
    double t;
    double dt = 0.001;
    double T = 1.0;

    int Nx = 40;
    int Ny = 40;
    int Npts = Nx*Ny;

    double dx = 1;
    double dy = 1;
    double Lx = dx*(Nx-1);
    double Ly = dy*(Ny-1);

    double nu = 0.1;

    double hnudt = nu*dt/dx/dx;

    int i, j;

    double pi = 2.0 * asin(1.0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double *h_A, *h_Anew; // CPU (host) vectors
    double *d_A, *d_Anew; // GPU(host) (device vectors)
    size_t size = Npts*sizeof(double); //size of each vector

    h_Anew = (double*)malloc(size); //Allocate CPU vectors
    h_A = (double*)malloc(size);

    cudaCheck( hipMalloc(&d_Anew, size));
    cudaCheck( hipMalloc(&d_A, size));

    for ( i = 0; i < Nx; i++) {
        h_A[IDX(i, 0)] = 0.0;
        h_Anew[IDX(i, 0)] = 0.0;
        h_A[IDX(i, Ny-1)] = 0.0;
        h_Anew[IDX(i, Ny-1)] = 0.0;
    }

    for ( j = 0; j < Ny; j++) {
        h_A[IDX(0, j)] = 0.0;
        h_Anew[IDX(0, j)] = 0.0;
        h_A[IDX(Nx-1, j)] = 0.0;
        h_Anew[IDX(Nx-1, j)] = 0.0;
    }

    for ( j = 1; j < Ny-1; j++) {
        for (i = 1;i < Nx-1;i++) {
            h_A[IDX(i,j)] = 5*sin(pi*(dx*i)/Lx)*sin(pi*(dy*j)/Ly);
            h_Anew[IDX(i,j)] = 5*sin(pi*(dx*i)/Lx)*sin(pi*(dy*j)/Ly);
        }
    }

    //Copy host vectors to device vector
    cudaCheck( hipMemcpy( d_Anew, h_Anew, size, hipMemcpyHostToDevice));
    cudaCheck( hipMemcpy( d_A, h_A, size, hipMemcpyHostToDevice));

    int blockSize, gridSize;
    blockSize = 1024; // One-dimensional block- number of threads
    gridSize = (int)ceil((double)Npts/blockSize); //One-d grid
                                                // number of grids

    hipEventRecord(start);
    for (t = 0; t < T; t+=dt) {
        // Execute the kernel
        interior_kernel<<<gridSize, blockSize>>>(d_Anew, d_A, Npts, Nx, hnudt);

        copy<<<gridSize, blockSize>>>(d_Anew, d_A, Npts);

        // Copy array back output vector from device to host
        cudaCheck( hipMemcpy( h_A, d_A, size, hipMemcpyDeviceToHost));
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // Sum up vector c and print result diveded by N: 1- no error
    double u;
    double error = 0.0;
    double max_error = 0.0;
    for( j = 1; j < Ny-1; j++ )
    {
      for(i = 1; i < Nx-1; i++)
      {
        u = 5*exp(-nu*pi*pi*(1/Lx/Lx+1/Ly/Ly)*T)*sin(pi/Lx*(dx*(i)))*sin(pi/Ly*(dy*(j)));

        error = error + sqrt(abs(h_A[IDX(i,j)]*h_A[IDX(i,j)]-u*u));
        max_error = fmax(max_error, fabs((h_A[IDX(i,j)]-u)/u));

        //std::cout << dx*(i) << ", "<< dy*(j) << ", " << (A[IDX(i,j)]) << ", " << u << ", " << fabs((A[IDX(i,j)]-u))/u << std::endl;
      }
    }

    printf("Max percentage error: %le\n", 100*max_error);
    printf("Runtime %fs\n ", milliseconds/10e2);
    cudaCheck( hipFree(d_Anew)); // Release device memory
    cudaCheck( hipFree(d_A));
    cudaCheck( hipFree(h_Anew)); // Release host memory
    cudaCheck( hipFree(h_A));
   

    return 0;



}